
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>
#include <math.h>

/* Program Parameters */
int N = 6000;
#define MAXN 6000
/* Matrices */
float *A, *B;
volatile float AA[MAXN][MAXN], BB[MAXN][MAXN];


/* Initialize A and B*/
void initialize_inputs() {
    int row, col;
    
    srand((unsigned)time(NULL));
    for (row = 0; row < N; row++) {
        for (col = 0; col < N; col++) {
            A[row*N+col] = (float)rand() / 32768.0;
            B[row*N+col] = 0.0;
            AA[row][col] = A[row*N+col];
            BB[row][col] = 0.0;
        }
    }
    
}


void print_output(){
    int r,c;
    printf("\nB =\n");
    for(r=0;r<N;r++){
        for(c=0;c<N;c++){
            printf("%5.5f%s", B[r*N+c], (c < N-1) ? ", " : ";\n");
        }
    }
}
void print_output2(){
    printf("\nBB =\n");
    int row, c;
    for(row=0;row<N;row++){
        for(c=0;c<N;c++){
             printf("%5.5f%s", BB[row][c], (c < N-1) ? ", " : ";\n");
        }
    }
}
void matrixNorm() {
    int row, col;
    float mu, sigma; // Mean and Standard Deviation
    
    for (col=0; col < N; col++) {
        mu = 0.0;
        for (row=0; row < N; row++)
            mu += AA[row][col];
        mu /= (float) N;
        sigma = 0.0;
        for (row=0; row < N; row++)
            sigma += powf(AA[row][col] - mu, 2.0);
        sigma /= (float) N;
        sigma = sqrt(sigma);
        for (row=0; row < N; row++) {
            if (sigma == 0.0)
                BB[row][col] = 0.0;
            else
                BB[row][col] = (AA[row][col] - mu) / sigma;
        }
    }
    
}


/* device function */


__global__ void matrixNorm(float *d_a, float *d_b, int n) {

    // get thread id(col) in the global scope
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    //printf("id:%d\n",col);
    int mu, sigma, row;
    mu = 0.0;
    // each threads load one element from global to shared mem.
    extern __shared__ float sdata[];
    
    for(row=0;row<n;row++){
        mu += d_a[row + col*n];
    }
    mu /= n;

    // make sure the calculation of mean is completed
    __syncthreads();

    sigma = 0.0;
    for (row=0; row < n; row++){
        sigma += powf(d_a[row+col*n] - mu, 2.0);
    }   
        sigma /= (float)n;
        sigma = sqrt(float(sigma));
    // make sure the calculation of standard deviation is completed
    __syncthreads();

    

    for(row=0;row<n;row++){
        if(sigma==0.0)
            d_b[row*n+col]=0.0;
        else
        d_b[row*n+col] = (d_a[row+col*n] - mu) / sigma;
    }

} 


int main(int argc, char **argv) {
    
    N = atoi(argv[1]);

    /* Initialize A and B */
    
    // allocate host memory for A, B
    A = (float*)malloc(4*N*N);
    B = (float*)malloc(4*N*N);

    
    initialize_inputs();

    // allocate device memory
    float *d_A, *d_B;
    hipMalloc((void**)&d_A, 4*N*N);
    hipMalloc((void**)&d_B, 4*N*N);

    printf("\n--------------------- CUDA Start------------------------\n");
    printf("Matrix size N = %d", N);
    printf("\nStarting clock.\n\n");

        /* Start Clock */

    // some events to count the execution time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);


    // start to count execution time of GPU version
    hipEventRecord(start,0);
    
    
    // copy the host data to device
    hipMemcpy((void*)d_A, (void*)A, 4*N*N, hipMemcpyHostToDevice);
    hipMemcpy((void*)d_B, (void*)B, 4*N*N, hipMemcpyHostToDevice);
    
    // set up dimension of grid and block, 2-dim gird and block
    dim3 blockSize(16);
    dim3 gridSize(ceil(N/((float) blockSize.x)));


    
    /* Matrix Normalization */
    
    matrixNorm<<<gridSize,blockSize>>>(d_A, d_B, N);
    
    // transfer result from device
    hipMemcpy(B, d_B, sizeof(float)*N*N, hipMemcpyDeviceToHost);
    hipEventRecord(stop,0);
    
    hipEventSynchronize(stop);

    // compute time elapse on GPU computing
    float gpu_elapsed_time_ms = 0.0;
    hipEventElapsedTime(&gpu_elapsed_time_ms, start, stop);
    printf("Time elapsed on GPU: %f ms.\n\n", gpu_elapsed_time_ms);
    printf("\nStopped clock.");
    print_output();
    printf("\n-------------------- CUDA End-------------------------\n");

        // // free both host and device memory
        free(A);
        free(B);
       hipFree(d_A);
       hipFree(d_B);
       
       

    // cpu serial computing
    struct timeval start2, stop2;  /* Elapsed times using gettimeofday() */
    struct timezone tzdummy;
    unsigned long long runtime2;
       /* Start Clock */
    printf("\n--------------------Serial Start-----------------------\n");
    printf("Matrix size N = %d", N);
    printf("\nStarting clock.\n\n");
    gettimeofday(&start2, &tzdummy);
    
    
    /* Matrix Normalization */
    matrixNorm();
    
    
    /* Stop Clock */
    gettimeofday(&stop2, &tzdummy);
    runtime2 = (unsigned long long)(stop2.tv_sec - start2.tv_sec) * 1000000 + (stop2.tv_usec - start2.tv_usec);
    
    
    /* Display timing results */
    printf("Runtime on CPU = %g ms.\n", (float)runtime2/(float)1000);
    printf("\nStopped clock.");
    print_output2();
    printf("\n---------------------Serial End---------------------------\n");
    
    
    exit(0);
}