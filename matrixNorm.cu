#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>
#include <math.h>

/* Program Parameters */
int N = 6000;

/* Matrices */
float *A, *B;


/* Initialize A and B*/
void initialize_inputs() {
    int row, col;
    
    srand((unsigned)time(NULL));
    for (row = 0; row < N; row++) {
        for (col = 0; col < N; col++) {
            A[row*N+col] = (float)rand() / 32768.0;
            B[row*N+col] = 0.0;
        }
    }
    
}


/* device function */


__global__ void matrixNorm(float *d_a, float *d_b, int n) {
    // get thread id(col) in the grid
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int mu, sigma, row;
    mu = 0.0;
    for(row=0;row<n;row++){
        mu += d_a[row*n + col];
    }
    mu /= n;

    // make sure the calculation of mean is completed
    __syncthreads();

    sigma = 0.0;
    for (row=0; row < n; row++){
        sigma += powf(d_a[row*n+col] - mu, 2.0);
    }   
        sigma /= n;
    // make sure the calculation of standard deviation is completed
    __syncthreads();

    sigma = sqrt(sigma);

    for(row=0;row<n;row++){
        if(sigma==0.0)
            d_b[row*n+col]=0.0;
        else
        d_b[row*n+col] = (d_a[row*n+col] - mu) / sigma;
    }

} 


int main(int argc, char **argv) {
    
    N = atoi(argv[1]);

    /* Initialize A and B */
    
    // allocate host memory for A, B
    A = (float*)malloc(4*N*N);
    B = (float*)malloc(4*N*N);

    initialize_inputs();

    // allocate device memory
    float *d_A, *d_B;
    hipMalloc((void**)&d_A, 4*N*N);
    hipMalloc((void**)&d_B, 4*N*N);

    
    // copy the host data to device
    hipMemcpy((void*)d_A, (void*)A, 4*N*N, hipMemcpyHostToDevice);
    hipMemcpy((void*)d_B, (void*)B, 4*N*N, hipMemcpyHostToDevice);
    
    // set up dimension of grid and block, 1-dim gird and block
    dim3 blockSize(32);
    dim3 gridSize((N+blockSize.x - 1) / blockSize.x);

    /* Start Clock */
    printf("\n---------------------------------------------\n");
    printf("Matrix size N = %d", N);
    printf("\nStarting clock.\n\n");
    float gpu_elapsed_time_ms;

    // some events to count the execution time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // start to count execution time of GPU version
    hipEventRecord(start, 0);
    
    
    /* Matrix Normalization */
    
    matrixNorm<<<gridSize,blockSize>>>(d_A, d_B, N);
    
    // transfer result from device
    hipMemcpy(B, d_B, sizeof(float)*N*N, hipMemcpyDeviceToHost);

    // free both host and device memory
    free(A);
    free(B);
    hipFree(d_A);
    hipFree(d_B);
    
    /* Stop Clock */
    hipDeviceSynchronize();
    // time counting terminate
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // compute time elapse on GPU computing
    hipEventElapsedTime(&gpu_elapsed_time_ms, start, stop);
    printf("Time elapsed on GPU: %f ms.\n\n", gpu_elapsed_time_ms);
    printf("\nStopped clock.");
    printf("\n---------------------------------------------\n");
    
    exit(0);
}