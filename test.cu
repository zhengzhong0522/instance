
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>
#include <math.h>


__global__ void cuda_hello(){
    printf("Hello World from GPU!\n");
}

int main() {
    
    cuda_hello<<<1,1>>>(); 
    printf("Hello World from CPU!\n");
    hipError_t err = hipGetLastError();
    if ( err != hipSuccess )
    {
       printf("CUDA Error: %s\n", hipGetErrorString(err));       

       // Possibly: exit(-1) if program cannot continue....
    }
    return 0;
}